#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

// Optimizations: #2, #4

#define Map_out_L1 4
#define Channel_L1 1
#define Map_out_L2 16
#define Channel_L2 4
#define MASK_WIDTH 7
#define TILE_WIDTH 16

static __constant__ float Mc1[Map_out_L1][Channel_L1*MASK_WIDTH*MASK_WIDTH];
static __constant__ float Mc2[Map_out_L2][Channel_L2*MASK_WIDTH*MASK_WIDTH];

__global__ void unroll_kernel(float *output, const float *input, const int Batch,
                              const int Channel, const int Height, const int Width,
                              const int K, const int start)
{
  const int Height_out = Height - K + 1;
  const int Width_out = Width - K + 1;

  int w = (blockIdx.x * blockDim.x) + threadIdx.x;

  #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
  #define unroll_3d(i2, i1, i0) output[(i2) * (Channel * K * K * Height_out * Width_out) + (i1) * (Height_out * Width_out) + i0]

  if (w < Batch * Height_out * Width_out) {
    int b = w / (Height_out * Width_out);
    w -= (b * Height_out * Width_out);
    int h = w / Width_out;
    w -= (h * Width_out);

    for (int c = 0; c < Channel; c++) {
      for (int p = 0; p < K; p++) {
        for (int q = 0; q < K; q++) {
          int col = (h * Width_out) + w;
          int row = (c * K * K) + (p * K) + q;
          unroll_3d(b, row, col) = in_4d(start+b, c, h + p, w + q);
        }
      }
    }
  }

  #undef in_4d
  #undef unroll_3d
}

__global__ void matrixMultiplyShared(float *output, const float *input,
                                     const int Batch, const int Map_out,
                                     const int Channel, const int Height,
                                     const int Width, const int K, const int start)
{
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int b = blockIdx.z;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0.0;

  const int inputSize = Channel*K*K;
  const int Height_out = Height - K + 1;
  const int Width_out = Width - K + 1;
  const int outputSize = Height_out * Width_out;

  #define input_3d(i2, i1, i0) input[(i2) * (Channel * K * K * Height_out * Width_out) + (i1) * (Height_out * Width_out) + i0]
  #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]

  for (int m = 0; m < ceil((1.0*inputSize)/TILE_WIDTH); ++m) {
    int subCol = (m * TILE_WIDTH) + tx;
    int subRow = (m * TILE_WIDTH) + ty;

    if (Row < Map_out && subCol < inputSize) {
      subTileA[ty][tx] = (Channel > 1) ? Mc2[Row][subCol] : Mc1[Row][subCol];
    }
    else {subTileA[ty][tx] = 0.0;}

    if (subRow < inputSize && Col < outputSize) {
      subTileB[ty][tx] = input_3d(b, subRow, Col);
    }
    else {subTileB[ty][tx] = 0.0;}

    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += subTileA[ty][k] * subTileB[k][tx];
    }
    __syncthreads();
  }

  if (Row < Map_out && Col < outputSize) {
    int h = Col / Width_out;
    int w = Col % Width_out;
    out_4d(start+b, Row, h, w) = Pvalue;
  }

  #undef input_3d
  #undef out_4d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int outputsize = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMalloc((void **) device_output_ptr, outputsize);

    int inputsize = Batch * Channel * Height * Width * sizeof(float);
    hipMalloc((void **) device_input_ptr, inputsize);
    hipMemcpy(*device_input_ptr, host_input, inputsize, hipMemcpyHostToDevice);

    // Copy Layer 1 or 2's kernel to constant memory
    int masksize = Map_out * Channel * K * K * sizeof(float);
    if (Channel > 1) {
      hipMemcpyToSymbol(HIP_SYMBOL(Mc2), host_mask, masksize);
    }
    else {
      hipMemcpyToSymbol(HIP_SYMBOL(Mc1), host_mask, masksize);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // Allocate memory space for the unrolled matrix
    float *unroll = NULL;
    int unroll_size = 1000 * Channel * K * K * Height_out * Width_out * sizeof(float);
    hipMalloc((void **)&unroll, unroll_size);

    for (int i = 0; i < ceil(Batch/1000.0); i++) {
      int start = i * 1000;
      int mini_b = min(1000, Batch - start);

      // Initialize grid and block dimensions here
      int numPixels = mini_b * Height_out * Width_out;
      unroll_kernel<<<ceil(numPixels/96.0),96>>>(unroll, device_input, Batch, Channel, Height, Width, K, start);
      hipDeviceSynchronize();

      dim3 DimGrid(ceil(Height_out * Width_out * 1.0 / TILE_WIDTH),
                   ceil(Map_out * 1.0 / TILE_WIDTH),
                   mini_b);
      dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
      matrixMultiplyShared<<<DimGrid,DimBlock>>>(device_output, unroll, Batch,
                                                 Map_out, Channel, Height, Width,
                                                 K, start);
      hipDeviceSynchronize();
    }

    hipFree(unroll);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMemcpy(host_output, device_output, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
