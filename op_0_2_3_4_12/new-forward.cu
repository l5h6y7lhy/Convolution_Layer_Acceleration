#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

// Optimizations: #0, #2, #3, #4, #12

#define Map_out_L1 4
#define Channel_L1 1
#define Map_out_L2 16
#define Channel_L2 4
#define MASK_WIDTH 7
#define T 128
#define U 16
#define S 8
#define SEG_SIZE 100

static __constant__ float Mc1[Map_out_L1][Channel_L1*MASK_WIDTH*MASK_WIDTH];
static __constant__ float Mc2[Map_out_L2][Channel_L2*MASK_WIDTH*MASK_WIDTH];


__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
  __shared__ float subTileN[S][U];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int b = blockIdx.z;
  int tx = threadIdx.x;

  int Col = bx * U;
  int Row = by * T;
  float Pvalue[U];

  for (int i = 0; i < U; i++) {Pvalue[i] = 0.0;}

  const int inputSize = Channel*K*K;
  const int Height_out = Height - K + 1;
  const int Width_out = Width - K + 1;
  const int outputSize = Height_out * Width_out;

  #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
  #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]

  for (int m = 0; m < ceil((1.0*inputSize)/S); ++m) {
    int subCol = Col + (tx % U);
    int subRow = (m * S) + (tx / U);

    if (subCol < Map_out && subRow < inputSize) {
      subTileN[tx / U][tx % U] = (Channel > 1) ? Mc2[subCol][subRow] : Mc1[subCol][subRow];
    }
    else {subTileN[tx / U][tx % U] = 0.0;}

    __syncthreads();

    for (int s = 0; s < S; s++) {
      subRow = Row + tx;
      subCol = (m * S) + s;

      float cw;

      if (subRow < outputSize && subCol < inputSize) {
        int c = subCol / (K * K);
        int p = subCol % (K * K);
        int q = p % K;
        p /= K;
        int h = subRow / Width_out;
        int w = subRow % Width_out;
        cw = in_4d(b, c, h+p, w+q);
      }
      else {cw = 0.0;}

      for (int i = 0; i < U; i++) {Pvalue[i] += cw * subTileN[s][i];}
    }

    __syncthreads();
  }

  for (int i = 0; i < U; i++) {
    if (Row + tx < outputSize && Col + i < Map_out) {
      int h = (Row + tx) / Width_out;
      int w = (Row + tx) % Width_out;
      out_4d(b, Col + i, h, w) = Pvalue[i];
    }
  }

  #undef in_4d
  #undef out_4d
}


// Calculate the number of remaining images after c epoches
__host__ int r(int c, const int Batch)
{
  return min(SEG_SIZE, Batch - (c * SEG_SIZE));
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
  // Allocate memory and copy over the relevant data structures to the GPU

  // We pass double pointers for you to initialize the relevant device pointers,
  //  which are passed to the other two functions.

  // Useful snippet for error checking
  // hipError_t error = hipGetLastError();
  // if(error != hipSuccess)
  // {
  //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
  //     exit(-1);
  // }

  // Copy Layer 1 or 2's kernel to constant memory
  int masksize = Map_out * Channel * K * K * sizeof(float);
  if (Channel > 1) {
    hipMemcpyToSymbol(HIP_SYMBOL(Mc2), host_mask, masksize);
  }
  else {
    hipMemcpyToSymbol(HIP_SYMBOL(Mc1), host_mask, masksize);
  }

  // Commonly used variables and macros
  const int Height_out = Height - K + 1;
  const int Width_out = Width - K + 1;
  int c = 0;
  int total = ceil(Batch*1.0/SEG_SIZE);

  int gx = ceil(Map_out * 1.0 / U);
  int gy = ceil(Height_out * Width_out * 1.0 / T);
  dim3 g;
  dim3 b = dim3(T, 1, 1);

  #define input_size(i) i * Channel * Height * Width
  #define output_size(i) i * Map_out * Height_out * Width_out

  // Start the three streams
  hipStream_t trans_in, compute, trans_out, tmp;
  hipStreamCreate(&trans_in);
  hipStreamCreate(&compute);
  hipStreamCreate(&trans_out);

  // Set up the input and output memory for the three streams
  float *input_ti;
  float *output_ti;
  float *input_c;
  float *output_c;
  float *input_to;
  float *output_to;
  float *dtmp;

  hipMalloc((void **)&input_ti, input_size(SEG_SIZE)*sizeof(float));
  hipMalloc((void **)&output_ti, output_size(SEG_SIZE)*sizeof(float));
  hipMalloc((void **)&input_c, input_size(SEG_SIZE)*sizeof(float));
  hipMalloc((void **)&output_c, output_size(SEG_SIZE)*sizeof(float));
  hipMalloc((void **)&input_to, input_size(SEG_SIZE)*sizeof(float));
  hipMalloc((void **)&output_to, output_size(SEG_SIZE)*sizeof(float));

  if (total <= 1 ) {
    // When the dataset is small
    hipMemcpyAsync(input_to, &host_input[0], input_size(Batch)*sizeof(float), hipMemcpyHostToDevice, trans_out);

    // Set up the kernel dimensions and zero the output memory (for atomicAdd)
    g = dim3(gx, gy, Batch);
    conv_forward_kernel<<<g,b,0,trans_out>>>(output_to, input_to, NULL, Batch, Map_out, Channel, Height, Width, K);
    hipMemcpyAsync(((void*) &host_output[0]), output_to, output_size(Batch)*sizeof(float), hipMemcpyDeviceToHost, trans_out);
  }
  else {
    // Start the first two streams
    hipMemcpyAsync(input_to, &host_input[0], input_size(SEG_SIZE)*sizeof(float), hipMemcpyHostToDevice, trans_out);
    c++;

    hipMemcpyAsync(input_c, &host_input[input_size(c*SEG_SIZE)], input_size(r(c, Batch)) * sizeof(float),
                    hipMemcpyHostToDevice, compute);
    g = dim3(gx, gy, SEG_SIZE);
    conv_forward_kernel<<<g,b,0,trans_out>>>(output_to, input_to, NULL, SEG_SIZE, Map_out, Channel, Height, Width, K);
    c++;

    while (c < total) {
      // One stream transfers its data into GPU
      // One stream does its computation work
      // One stream transfers its data out of GPU
      hipMemcpyAsync(input_ti, &host_input[input_size(c*SEG_SIZE)], input_size(r(c, Batch)) * sizeof(float),
                      hipMemcpyHostToDevice, trans_in);
      conv_forward_kernel<<<g,b,0,compute>>>(output_c, input_c, NULL, SEG_SIZE, Map_out, Channel, Height, Width, K);
      hipMemcpyAsync(((void*) &host_output[output_size((c-2)*SEG_SIZE)]), output_to, output_size(SEG_SIZE)*sizeof(float),
                      hipMemcpyDeviceToHost, trans_out);

      // Swap the three streams and their memory pointers
      tmp = compute;
      compute = trans_in;
      trans_in = trans_out;
      trans_out = tmp;

      dtmp = input_c;
      input_c = input_ti;
      input_ti = input_to;
      input_to = dtmp;

      dtmp = output_c;
      output_c = output_ti;
      output_ti = output_to;
      output_to = dtmp;

      c++;
    }

    g = dim3(gx, gy, r(c - 1, Batch));

    // Wrap up the computation and transferring work for two streams
    conv_forward_kernel<<<g,b,0,compute>>>(output_c, input_c, NULL, r(c-1, Batch), Map_out, Channel, Height, Width, K);
    hipMemcpyAsync(((void*) &host_output[output_size((c-2)*SEG_SIZE)]), output_to, output_size(SEG_SIZE)*sizeof(float),
                    hipMemcpyDeviceToHost, trans_out);
    hipMemcpyAsync(((void*) &host_output[output_size((c-1)*SEG_SIZE)]), output_c, output_size(r(c-1, Batch)) * sizeof(float),
                    hipMemcpyDeviceToHost, compute);
  }

  // Make sure all of the three streams have finished all their work in the queue
  hipStreamSynchronize(trans_out);
  hipStreamSynchronize(compute);
  hipStreamSynchronize(trans_in);

  hipStreamDestroy(trans_out);
  hipStreamDestroy(compute);
  hipStreamDestroy(trans_in);

  // Free memory space
  hipFree(input_ti);
  hipFree(output_ti);
  hipFree(input_c);
  hipFree(output_c);
  hipFree(input_to);
  hipFree(output_to);

  #undef input_size
  #undef output_size
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    return;
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    return;
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
